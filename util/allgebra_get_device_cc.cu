#include <stdio.h>
#include "hip/hip_runtime.h"

int main()
{
	hipDeviceProp_t prp;
	hipError_t cudaStatus;

	cudaStatus = hipGetDeviceProperties(&prp, /*device=*/0);
	if (cudaStatus != hipSuccess) {
		printf("hipGetDeviceProperties error");
		return 1;
	}

	printf("%d%d\n", prp.major, prp.minor);

	return 0;
}
