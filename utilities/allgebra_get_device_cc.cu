#include <stdio.h>
#include "hip/hip_runtime.h"

int main()
{
	hipDeviceProp_t prop;
	hipError_t cudaStatus;

	cudaStatus = hipGetDeviceProperties(&prop, 0); // 0 is device number
	if (cudaStatus != hipSuccess) {
		printf("hipGetDeviceProperties error");
		return 1;
	}

	printf("%d%d\n", prop.major, prop.minor);

	return 0;
}
