/**
 * Copyright 2020 RICOS Co. Ltd.
 *
 * This file is a part of ricosjp/allgebra, distributed under Apache-2.0 License
 * https://github.com/ricosjp/allgebra
 */

#include <stdio.h>
#include "hip/hip_runtime.h"

int main()
{
  hipDeviceProp_t prop;
  hipError_t cudaStatus;

  cudaStatus = hipGetDeviceProperties(&prop, 0); // 0 is device number
  if (cudaStatus != hipSuccess) {
    return 1;
  }
  printf("%d%d\n", prop.major, prop.minor);
  return 0;
}
